
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

// function to add the elements of two arrarys
__global__
void add(int n, float *x, float *y)
{

  int index = threadIdx.x;
  int stride = blockDim.x;
  printf("%d Index, %d blockDim, %d blockIdx\n", index, stride,blockIdx.x);
  for(int i = index; i < n ; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;

  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for(int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  add<<<1, 256>>>(N, x, y);

  hipDeviceSynchronize();
  float maxError = 0.0f;
  for(int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  //free
  //delete [] x;
  //delete [] y;
  hipFree(x);
  hipFree(y);

  return 0;
}
