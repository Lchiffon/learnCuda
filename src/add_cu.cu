
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrarys
__global__
void add(int n, float *x, float *y)
{
  for(int i = 0; i < n ; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;

  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for(int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  add<<<1, 1>>>(N, x, y);

  hipDeviceSynchronize();
  float maxError = 0.0f;
  for(int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  //free
  //delete [] x;
  //delete [] y;
  hipFree(x);
  hipFree(y);

  return 0;
}
